#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include "kernel.h"
#include "invoke.h"
#include <cassert>
#include <iostream>
#include <limits>
#define SIZE 1024
#define Block 1
#define FULL_WARP_MASK 0xFFFFFFFF


template <class T>
__device__ T warp_reduce(T val){
    for(int offset=32/2;offset>0;offset/=2)
        val+= __shfl_down_sync (FULL_WARP_MASK,val,offset);
    return val;

}

typedef float (*op_scalar_fn)(float, float);

__device__ inline float add_scalar(float x, float y) {
    return x + y;
}

__device__ inline float sub_scalar(float x, float y) {
    return x - y;
}

__device__ inline float max_scalar(float x, float y) {
    if(x>y) return x;
    else return y;
}

__device__ inline float min_scalar(float x, float y) {
    if(x<y) return x;
    else return y;
}

__device__ inline float mul_scalar(float x, float y) {
    return x * y;
}

__device__ inline float div_scalar(float x, float y) {
    return x / y;
}

__device__ op_scalar_fn  p_mul = mul_scalar;
__device__ op_scalar_fn  p_div = div_scalar;
__device__ op_scalar_fn  p_add = add_scalar;
__device__ op_scalar_fn  p_sub = sub_scalar;
__device__ op_scalar_fn  p_min = min_scalar;
__device__ op_scalar_fn  p_max = max_scalar;

//to be used if host is sending function pointer to kernel
inline op_scalar_fn get_fn(op_t op) {
    op_scalar_fn op_fn;

    if (op == eDIV) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_div), sizeof(op_scalar_fn));
        //op_fn = div_scalar;
    } else if (op == eSUB) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_sub), sizeof(op_scalar_fn));
        //op_fn = sub_scalar;
    } else if (op == eSUM) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_add), sizeof(op_scalar_fn));
        //op_fn = add_scalar;
    } else if (op == eMUL) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_mul), sizeof(op_scalar_fn));
        //op_fn = mul_scalar;
    } else if (op == eMIN) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_min), sizeof(op_scalar_fn));
        //op_fn = min_scalar;
    } else if (op == eMAX) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_max), sizeof(op_scalar_fn));
        //op_fn = max_scalar;
    } else {
        assert(0);
    }
    return op_fn;
}

//if the kernel itself need the fuction pointer
__device__ inline op_scalar_fn get_fn_kernel(op_t op) {
    op_scalar_fn op_fn;

    if (op == eDIV) {
        op_fn = div_scalar;
    } else if (op == eSUB) {
        op_fn = sub_scalar;
    } else if (op == eSUM) {
        op_fn = add_scalar;
    } else if (op == eMUL) {
        op_fn = mul_scalar;
    } else if (op == eMIN) {
        op_fn = min_scalar;
    } else if (op == eMAX) {
        op_fn = max_scalar;
    } else {
        assert(0);
    }
    return op_fn;
}

__global__ void spmm(const csr_t* __restrict__ obj1, float* x, float * y, op_t op, const bool reverse, const bool norm, const int dim) 
{
    int row_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row_index >= obj1->v) {
        return;
    }

    vid_t degree;
    float * input = x + dim * row_index;
    float * output = y + dim * row_index;
    vid_t * nebrs;
    degree = obj1->get_nebrs(row_index, nebrs);

     // Normalize the input
    if (reverse) {
        for (vid_t i = 0 ; i < dim ; ++i) {
            input[i] /= degree + 1;
        }
    }

    // Self-loop
    for (vid_t i = 0 ; i < dim ; i++) {
        output[i] = input[i];
    }

    // Multiplication
    for (vid_t j = 0 ; j < degree ; j++) {
        float * nebrs_ptr = x + dim * nebrs[j];
        for (vid_t k = 0 ; k < dim ; ++k) {
            output[k] += nebrs_ptr[k];
        }
    }

    // Normalize the output
    if (!reverse) {
        for (vid_t i = 0 ; i < dim ; ++i) {
            output[i] /= degree + 1;
        }
    }
}

//warp per row (best)
__global__ void spmm_warp(const csr_t* __restrict__ obj1, float* x, float * y, op_t op, const bool reverse, const bool norm, const int dim)
{
    //TODO  
}

void invoke_spmm(csr_t * obj1, array2d_t < float > & x1, array2d_t < float > & y1, op_t op, bool reverse, bool norm, int dim) {
    int warp_size=32;
    int block_size=1024;

    // spmm
    int nBlocks =  (int) ceil(obj1->v / (float)(block_size));; // TODO 
    spmm <<<nBlocks,block_size>>> (obj1, x1.data_ptr, y1.data_ptr, op, reverse, true, dim);

    // spmm_warp
    //int nBlocks =  (int) ceil(obj1->v / (float)(block_size / warp_size));; // TODO 
    //spmm_warp <<<nBlocks,block_size>>> (obj1, x1.data_ptr, y1.data_ptr, op, reverse, true, dim);

    hipDeviceSynchronize();
}

graph_t * invoke_init_graph(vid_t v_count, vid_t dst_size, vid_t * offset_csr, void * nebrs_csr, vid_t * offset_csc, void * nebrs_csc) {

    //Let us make a cpu graph first
    graph_t g;
    g.init_cpu(v_count, dst_size, 
            offset_csr, nebrs_csr,
            offset_csc, nebrs_csc);

    graph_t * graph = (graph_t*) malloc(sizeof(graph_t));
    hipMallocManaged( & graph->csr,  sizeof(csr_t));

    vid_t edge_count = offset_csr[v_count];
    vid_t * offset_csr_gpu;
    vid_t * offset_csc_gpu;
    char * nebrs_csr_gpu;
    char * nebrs_csc_gpu;

    hipMallocManaged( & offset_csr_gpu, (v_count + 1) * sizeof(vid_t));
    hipMallocManaged( & nebrs_csr_gpu, edge_count * dst_size);

    //memcopy
    hipMemcpy(offset_csr_gpu, offset_csr, (v_count + 1) * sizeof(vid_t), hipMemcpyHostToDevice);
    hipMemcpy(nebrs_csr_gpu, nebrs_csr, edge_count * dst_size, hipMemcpyHostToDevice);


    if (nebrs_csr == nebrs_csc) {
        graph->csc = graph->csr;
        offset_csc_gpu = offset_csr_gpu;
        nebrs_csc_gpu = nebrs_csr_gpu;
    } else {
        hipMallocManaged( & graph->csc,  sizeof(csr_t));
        hipMallocManaged( & offset_csc_gpu, (v_count + 1) * sizeof(vid_t));
        hipMallocManaged( & nebrs_csc_gpu, edge_count * dst_size);

        hipMemcpy(nebrs_csc_gpu, nebrs_csc, edge_count * dst_size, hipMemcpyHostToDevice);
        hipMemcpy(offset_csc_gpu, offset_csc, (v_count + 1) * sizeof(vid_t), hipMemcpyHostToDevice);
    }

    //printf("invoke init graph called\n");
    graph -> init(v_count, dst_size, offset_csr_gpu, nebrs_csr_gpu, offset_csc_gpu, nebrs_csc_gpu);

    return graph;

}

